#include "hip/hip_runtime.h"
#include "reduction.h"
#include "tracing.h"
#include <thrust/device_vector.h>
#include "memoryWrapper.cuh"
#include <>
#include <cassert>
#include <iostream>
#include "reduction.cuh"

float thrust_reduce(std::vector<float>const& in, size_t n_iter)
{
    Trace::set(tracepoint::start);

    Trace::set(tracepoint::copy_start);
    thrust::device_vector<float>(in.begin(),in.end());
    Trace::set(tracepoint::copy_end);
    float sum = 0.f;
    for(size_t i=0; i!= n_iter;++i)
        sum = thrust::reduce(in.begin(), in.end());

    Trace::set(tracepoint::backcopy_start);
    Trace::set(tracepoint::backcopy_end);
    Trace::set(tracepoint::end);
    return sum;
}


template<typename T>
__global__ void reduce_kernel_optim(T* __restrict volatile in, T* __restrict out)
{
    // So just putting "T smem[]" here is too easy for cuda. Need to hand-cast it to the type we want.
    //FML. https://stackoverflow.com/questions/27570552/templated-cuda-kernel-with-dynamic-shared-memory
    // By the way, it first was just a float which of course doesn't really work for the other types and caused
    // some kind of stack corruption where every subsequent kernel would just go bananas
    extern __shared__ unsigned char evil_smem[];
    T* smem = reinterpret_cast<T*>(evil_smem);


    auto const tid_glob = threadIdx.x + blockIdx.x * blockDim.x;
    auto const tid_loc = threadIdx.x;
    auto const n_total_threads = blockDim.x * gridDim.x;

    smem[tid_loc] = in[tid_glob+n_total_threads];
    smem[tid_loc] += in[tid_glob];

    //now all we care about is the memory block that corresponds to our block size

    for(uint32_t threads_alive = blockDim.x/2;
    threads_alive>32 && threadIdx.x < threads_alive;
    threads_alive>>=1)
    {
        __syncthreads();
        smem[tid_loc] += smem[tid_loc+threads_alive];
    }
    if(threadIdx.x < 32 && blockDim.x > 64) {
        smem[tid_loc] += smem[tid_loc + threadIdx.x];
    }
    if(threadIdx.x < 16 && blockDim.x > 32) {
        smem[tid_loc] += smem[tid_loc + threadIdx.x];
    }
    if(threadIdx.x < 8 && blockDim.x > 16) {
        smem[tid_loc] += smem[tid_loc + threadIdx.x];
    }
    if(threadIdx.x < 4 && blockDim.x > 8) {
        smem[tid_loc] += smem[tid_loc + threadIdx.x];
    }
    if(threadIdx.x < 2 && blockDim.x > 4) {
        smem[tid_loc] += smem[tid_loc + threadIdx.x];
    }
    if(threadIdx.x < 1 && blockDim.x > 2) {
        smem[tid_loc] += smem[tid_loc + threadIdx.x];
    }


    if (threadIdx.x == 0)
        out[blockIdx.x] = smem[tid_loc];
}


template<typename T>
T reduce_cuda_optim(std::vector<T>& in, uint32_t const n_blocks, size_t iters)
{
    Trace::set(tracepoint::start);
    //zero pad end of vector if it doesn't fit
    if(!is_power_of_2(in.size()))
    {
        uint32_t next_power = ceil(log2((double)in.size()));
        uint32_t new_size = pow(2,next_power);
        in.resize(new_size, 0);
    }
    uint32_t const threads_total = in.size()/2;
    uint32_t const threads_per_block = threads_total / n_blocks;
    uint32_t const shared_size  = threads_per_block * sizeof(T);

    assert(threads_per_block <= 1024);
    assert(is_power_of_2(threads_per_block));
    assert(is_power_of_2(n_blocks));

    Trace::set(tracepoint::copy_start);
    DeviceMemory<T> d_in(in.data(), in.size());
    Trace::set(tracepoint::copy_end);
    DeviceMemory<T> d_out(n_blocks);
    DeviceMemory<T> d_final_out(1);

    //output is a single value per block
    for(auto i=0;i!=iters;++i){
        reduce_kernel_optim<T><<<n_blocks,threads_per_block, shared_size>> > (d_in.mem(), d_out.mem());
        hipDeviceSynchronize();
    }
    throwOnCudaError();

    std::vector<T> result(0);

    //use a single block with n_blocks threads to do final summation
    if(n_blocks>1) {
        auto const remaining_threads = n_blocks/2;
        auto const remaining_memsize = remaining_threads * sizeof(T);

        reduce_kernel_optim<T> << < 1, remaining_threads, remaining_memsize >> > (d_out.mem(), d_final_out.mem());
        Trace::set(tracepoint::backcopy_start);
        result = d_final_out.to_vector();
        Trace::set(tracepoint::backcopy_end);
    }
        //unless we're already done;
    else{
        result.push_back(d_out.to_vector()[0]);
    }

    assert(result.size() == 1);
    Trace::set(tracepoint::end);
    return result[0];
}

float reduce_cuda_optim(std::vector<float>& in, uint32_t const n_blocks, size_t iters)
{
    return reduce_cuda_optim<float>(in, n_blocks, iters);
}