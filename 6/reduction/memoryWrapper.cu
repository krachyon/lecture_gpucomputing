#include "memoryWrapper.cuh"
#include <cassert>


void hipMemcpy(Memory const& dest, Memory const& src) {
    assert(dest.size >= src.size);

    hipMemcpyKind kind = hipMemcpyDefault;

    if (dest.kind == memKind::device) {
        if (src.kind == memKind::device)
            kind = hipMemcpyDeviceToDevice;
        else if (src.kind == memKind::pinned || src.kind == memKind::host)
            kind = hipMemcpyHostToDevice;
        else
            throw (std::logic_error{"unknown memory type encountered"});
    } else if (dest.kind == memKind::pinned || dest.kind == memKind::host) {
        if (src.kind == memKind::device)
            kind = hipMemcpyDeviceToHost;
        else if (src.kind == memKind::pinned || src.kind == memKind::host)
            kind = hipMemcpyHostToHost;
        else
            throw (std::logic_error{"unknown memory type encountered"});
    }
    checkCuda(hipMemcpy(dest._mem, src._mem, dest.size, kind));
}

