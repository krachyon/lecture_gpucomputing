#include "hip/hip_runtime.h"
#include "reduction.cuh"
#include "tracing.h"
#include "memoryWrapper.cuh"
#include <>
#include <cassert>


template<typename T>
__global__ void reduce_kernel_shared(T* __restrict volatile in, T* __restrict out)
{
    extern __shared__ float smem[];
    auto const tid_glob = threadIdx.x + blockIdx.x * blockDim.x;
    auto const tid_loc = threadIdx.x;
    auto const n_total_threads = blockDim.x * gridDim.x;

    uint32_t threads_alive = blockDim.x;

    smem[tid_loc] = in[tid_glob+n_total_threads];
    smem[tid_loc] += in[tid_glob];

    threads_alive >>= 1;
    //now all we care about is the memory block that corresponds to our block size

    while(threads_alive!=0 && threadIdx.x < threads_alive)
    {
        __syncthreads();
        smem[tid_loc] += smem[tid_loc+threads_alive];
        threads_alive >>= 1;
    }
    __syncthreads();

    if (threadIdx.x == 0)
        out[blockIdx.x] = smem[tid_loc];

}


template<typename T>
T reduce_cuda_shared(std::vector<T>& in, uint32_t const n_blocks)
{
    Trace::set("cuda_shared_start");
    //zero pad end of vector if it doesn't fit
    if(!is_power_of_2(in.size()))
    {
        uint32_t next_power = ceil(log2((double)in.size()));
        uint32_t new_size = pow(2,next_power);
        in.resize(new_size, 0);
    }
    uint32_t const threads_total = in.size()/2;
    uint32_t const threads_per_block = threads_total / n_blocks;
    uint32_t const shared_size  = threads_per_block * sizeof(T);

    assert(threads_per_block <= 1024);
    assert(is_power_of_2(threads_per_block));
    assert(is_power_of_2(n_blocks));

    Trace::set("cuda_shared_copy_in");
    DeviceMemory<T> d_in(in.data(), in.size());
    Trace::set("cuda_shared_copy_in_done");
    DeviceMemory<T> d_out(n_blocks);
    DeviceMemory<T> d_final_out(1);

    //output is a single value per block
    reduce_kernel_shared<T><<<n_blocks,threads_per_block, shared_size>> > (d_in.mem(), d_out.mem());
    hipDeviceSynchronize();

    std::vector<T> result(0);

    //use a single block with n_blocks threads to do final summation
    if(n_blocks>1) {
        reduce_kernel_shared<T> << < 1, n_blocks / 2 >> > (d_out.mem(), d_final_out.mem());
        Trace::set("cuda_shared_copy_out");
        result = d_final_out.to_vector();
        Trace::set("cuda_shared_copy_out_done");
    }
        //unless we're already done;
    else{
        result.push_back(d_out.to_vector()[0]);
    }

    assert(result.size() == 1);
    Trace::set("cuda_shared_end");
    return result[0];
}

#include <thrust/device_vector.h>

float thrust_reduce(std::vector<float>const& in)
{
    thrust::device_vector<float>(in.begin(),in.end());
    float sum = thrust::reduce(in.begin(), in.end());
    return sum;
}

