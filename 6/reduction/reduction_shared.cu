#include "hip/hip_runtime.h"
#include "reduction.cuh"
#include "tracing.h"
#include "memoryWrapper.cuh"
#include <>
#include <cassert>
#include <iostream>

template<typename T>
__global__ void reduce_kernel_shared(T* __restrict volatile in, T* __restrict out)
{
    // So just putting "T smem[]" here is too easy for cuda. Need to hand-cast it to the type we want.
    //FML. https://stackoverflow.com/questions/27570552/templated-cuda-kernel-with-dynamic-shared-memory
    // By the way, it first was just a float which of course doesn't really work for the other types and caused
    // some kind of stack corruption where every subsequent kernel would just go bananas
    extern __shared__ unsigned char evil_smem[];
    T* smem = reinterpret_cast<T*>(evil_smem);


    auto const tid_glob = threadIdx.x + blockIdx.x * blockDim.x;
    auto const tid_loc = threadIdx.x;
    auto const n_total_threads = blockDim.x * gridDim.x;

    uint32_t threads_alive = blockDim.x;

    smem[tid_loc] = in[tid_glob+n_total_threads];
    smem[tid_loc] += in[tid_glob];

    threads_alive >>= 1;
    //now all we care about is the memory block that corresponds to our block size

    while(threads_alive!=0 && threadIdx.x < threads_alive)
    {
        __syncthreads();
        smem[tid_loc] += smem[tid_loc+threads_alive];
        threads_alive >>= 1;
    }
    __syncthreads();

    if (threadIdx.x == 0)
        out[blockIdx.x] = smem[tid_loc];

}


template<typename T>
T reduce_cuda_shared(std::vector<T>& in, uint32_t const n_blocks)
{
    Trace::set("cuda_shared_start");
    //zero pad end of vector if it doesn't fit
    if(!is_power_of_2(in.size()))
    {
        uint32_t next_power = ceil(log2((double)in.size()));
        uint32_t new_size = pow(2,next_power);
        in.resize(new_size, 0);
    }
    uint32_t const threads_total = in.size()/2;
    uint32_t const threads_per_block = threads_total / n_blocks;
    uint32_t const shared_size  = threads_per_block * sizeof(T);

    assert(threads_per_block <= 1024);
    assert(is_power_of_2(threads_per_block));
    assert(is_power_of_2(n_blocks));

    Trace::set("cuda_shared_copy_in");
    DeviceMemory<T> d_in(in.data(), in.size());
    Trace::set("cuda_shared_copy_in_done");
    DeviceMemory<T> d_out(n_blocks);
    DeviceMemory<T> d_final_out(1);

    //output is a single value per block
    reduce_kernel_shared<T><<<n_blocks,threads_per_block, shared_size>> > (d_in.mem(), d_out.mem());
    hipDeviceSynchronize();
    throwOnCudaError();

    std::vector<T> result(0);

    //use a single block with n_blocks threads to do final summation
    if(n_blocks>1) {
        auto const remaining_threads = n_blocks/2;
        auto const remaining_memsize = remaining_threads * sizeof(T);

        reduce_kernel_shared<T> << < 1, remaining_threads, remaining_memsize >> > (d_out.mem(), d_final_out.mem());
        Trace::set("cuda_shared_copy_out");
        result = d_final_out.to_vector();
        Trace::set("cuda_shared_copy_out_done");
    }
        //unless we're already done;
    else{
        result.push_back(d_out.to_vector()[0]);
    }

    assert(result.size() == 1);
    Trace::set("cuda_shared_end");
    return result[0];
}

float reduce_cuda_shared(std::vector<float>& in, uint32_t const n_blocks)
{
    return reduce_cuda_shared <float>(in, n_blocks);
}
double reduce_cuda_shared(std::vector<double>& in, uint32_t const n_blocks)
{
    return reduce_cuda_shared<double>(in, n_blocks);
}
uint32_t reduce_cuda_shared(std::vector<uint32_t>& in, uint32_t const n_blocks)
{
    return reduce_cuda_shared<uint32_t>(in, n_blocks);
}
int32_t reduce_cuda_shared(std::vector<int32_t>& in, uint32_t const n_blocks)
{
    return reduce_cuda_shared<int32_t>(in, n_blocks);
}
int16_t reduce_cuda_shared(std::vector<int16_t>& in, uint32_t const n_blocks)
{
    return reduce_cuda_shared<int16_t>(in, n_blocks);
}
uint16_t reduce_cuda_shared(std::vector<uint16_t>& in, uint32_t const n_blocks)
{
    return reduce_cuda_shared<uint16_t>(in, n_blocks);
}



//TODO move me
#include <thrust/device_vector.h>
float thrust_reduce(std::vector<float>const& in)
{
    thrust::device_vector<float>(in.begin(),in.end());
    float sum = thrust::reduce(in.begin(), in.end());
    return sum;
}

