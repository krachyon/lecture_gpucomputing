#include "hip/hip_runtime.h"
#include "reduction.cuh"

#include "memoryWrapper.cuh"

#include <>
#include <cassert>
//#include <cmath>

//TODO replace log2/ceil with integer math

template<typename T>
__device__ __host__ bool is_power_of_2(T in)
{
    return log2((double)in) == floor(log2((double)in));
};

template<typename T>
__global__ void reduce_kernel_naive(T* __restrict volatile in, T* __restrict out)
//bytes(out) == gridDim.x
{
    auto const tid = threadIdx.x + blockIdx.x * blockDim.x;
    auto const n_total_threads = blockDim.x * gridDim.x;

    //ensure that threadblock has a power of 2 as bytes

    if(tid==0)
    {
        for(auto i =0;i!=n_total_threads*2;++i)
        printf("%f ", in[i]);
        printf("\n");
    }
    printf("thread %i before loop\n",threadIdx.x);

    uint32_t threads_alive = blockDim.x;

    // first computation: Fetch elements on right side of memory
    in[tid] += in[tid+n_total_threads];
    threads_alive >>= 1;
    //now all we care about is the memory block that corresponds to our block size

    __syncthreads();
    printf("[%i] += [%i]\n",tid, tid+n_total_threads);

    while(threads_alive!=0 && threadIdx.x < threads_alive)
    {
        __syncthreads();
        printf("threads alive %i\n", threads_alive);
        printf("%f [%i] += %f [%i]\n", in[tid], tid, in[tid+threads_alive], tid+threads_alive);
        __syncthreads();

        in[tid] += in[tid+threads_alive];
        threads_alive >>= 1;

        __syncthreads();
    }
    printf("thread %i after loop\n",threadIdx.x);

    if(threadIdx.x != 0)
        return;
    else if (threadIdx.x == 0)
    {
        __syncthreads();
        printf("result %f\n",in[tid]);
        // We are the last thread in the block; "report" the result
        out[blockIdx.x] = in[tid];
        return;
    }
}

template<typename T>
T reduce_cuda_naive(std::vector<T>& in, uint32_t const n_blocks)
{
    //zero pad end of vector if it doesn't fit
    if(!is_power_of_2(in.size()))
    {
        uint32_t next_power = ceil(log2((double)in.size()));
        uint32_t new_size = pow(2,next_power);
        in.resize(new_size, 0);
    }
    uint32_t threads_total = in.size()/2;
    uint32_t threads_per_block = threads_total / n_blocks;

    assert(threads_per_block <= 1024);
    assert(is_power_of_2(threads_per_block));
    assert(is_power_of_2(n_blocks));

    DeviceMemory<T> d_in(in.data(), in.size());
    DeviceMemory<T> d_out(n_blocks);
    DeviceMemory<T> d_final_out(1);

    //output is a single value per block
    reduce_kernel_naive<T><<<n_blocks,threads_per_block>> > (d_in.mem(), d_out.mem());
    hipDeviceSynchronize();
    //use a single block with n_blocks threads to do final summation
    //TODO this does not currently work with a single block in the first iteration as the kernel will do a first summation
    // regardless. See if this can be adapted
    reduce_kernel_naive<T><< < 1, n_blocks/2 >> > (d_out.mem(),d_final_out.mem());

    auto result = d_final_out.to_vector();
    assert(result.size() == 1);
    return result[0];
}

//relevant implementations of reduction.h

float reduce_cuda_naive(std::vector<float>& in, uint32_t const n_blocks)
{
    return reduce_cuda_naive<float>(in, n_blocks);
}
