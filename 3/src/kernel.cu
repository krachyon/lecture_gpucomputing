
#include <hip/hip_runtime.h>
/*************************************************************************************************
 *
 *        Computer Engineering Group, Heidelberg University - GPU Computing Exercise 03
 *
 *                           Group : TBD
 *
 *                            File : main.cu
 *
 *                         Purpose : Memory Operations Benchmark
 *
 *************************************************************************************************/

//
// Kernels
//

__global__ void 
globalMemCoalescedKernel(/*TODO Parameters*/)
{
    /*TODO Kernel Code*/
}

void 
globalMemCoalescedKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemCoalescedKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

__global__ void 
globalMemStrideKernel(/*TODO Parameters*/)
{
    /*TODO Kernel Code*/
}

void 
globalMemStrideKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemStrideKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

__global__ void 
globalMemOffsetKernel(/*TODO Parameters*/)
{
    /*TODO Kernel Code*/
}

void 
globalMemOffsetKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemOffsetKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

