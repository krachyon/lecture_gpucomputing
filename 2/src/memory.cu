
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <chrono>
#include <iostream>
#include <vector>
using std::chrono::high_resolution_clock;
using std::chrono::nanoseconds;


// Quick and dirty RAII wrappers to avoid memleaks
struct Memory
{
    Memory() = default;
    Memory(Memory const&) = delete;
    Memory(Memory &&) = delete;
    Memory& operator=(Memory const&) = delete;
    Memory& operator=(Memory &&) = delete;
    virtual ~Memory(){};


    void* _mem;
    size_t size;
};

struct DeviceMemory: public Memory
{
    DeviceMemory(size_t count)
    {
        size = count*sizeof(uint8_t);
        hipMalloc(&_mem, size);
    }
    virtual ~DeviceMemory(){hipFree(_mem);}
};

struct HostMemory: public Memory
{
    HostMemory(size_t count)
    {
        size = count*sizeof(uint8_t);
        _mem = malloc(size);
    }
    virtual ~HostMemory(){free(_mem);}
};

struct PinnedMemory: public Memory
{
    PinnedMemory(size_t count)
    {
        size = count*sizeof(uint8_t);
        hipHostMalloc(&_mem, size, hipHostMallocDefault);
    }
    virtual ~PinnedMemory(){hipHostFree(_mem);}
};

template <typename HostMem>
size_t timeDeviceToHost(size_t count)
{
    DeviceMemory dev(count);
    HostMem host(count);

    auto start = high_resolution_clock::now();
    hipMemcpy(dev._mem, host._mem, host.size, hipMemcpyDeviceToHost);
    auto end = high_resolution_clock::now();
    return std::chrono::duration_cast<nanoseconds>(end-start).count();
}

template <typename HostMem>
size_t timeHostToDevice(size_t count)
{
    DeviceMemory dev(count);
    HostMem host(count);

    auto start = high_resolution_clock::now();
    hipMemcpy(host._mem, dev._mem, dev.size, hipMemcpyHostToDevice);
    auto end = high_resolution_clock::now();
    return std::chrono::duration_cast<nanoseconds>(end-start).count();
}

int main()
{
    size_t kb = 1024;
    size_t GB = kb*kb*kb;
    std::vector<size_t> sizes;
    for(size_t current = kb; current <= GB; current*=2)
        sizes.push_back(current);

    std::cout << "#Size; H2DPage; H2DPin; D2HPage; D2HPin" << std::endl;
    for(auto size: sizes)
    {
        std::cout << size << ";"
         << timeHostToDevice<HostMemory>(size) << ";"
         << timeHostToDevice<PinnedMemory>(size) << ";"
         << timeDeviceToHost<HostMemory>(size) << ";"
         << timeDeviceToHost<PinnedMemory>(size) << std::endl;
    }
}
