#include "hip/hip_runtime.h"
#include "nbody.h"
#include "utility.cuh"
#include "errorHandling.cuh"

#include <cstdint>
#include <cstddef>
#include <>
#include <random>

template<typename Body_t>
__device__ float3 accel(Body_t* bodies, size_t idx, size_t N)
{
    float3 dv{0,0,0};
    Body_t const& body = bodies[idx];

    for(size_t i=0; i!=N; ++i)
    {
        if(i==idx)
            continue;
        Body_t const& current_body = bodies[i];

        auto diff = current_body.pos - body.pos;
        dv += G * current_body.m * diff / (norm_pow3(diff) + eps);
    }

    return dv;
}

template<typename Body_t>
__global__ void leapfrog_aos(Body_t* bodies, size_t N, size_t iters)
{
    uint32_t tid = threadIdx.x + gridDim.x * blockIdx.x;
    if(tid>=N)
        return;

    float3 vel_half = bodies[tid].vel + accel(bodies,tid,N) * dt/2;
    bodies[tid].pos += vel_half * dt;
    vel_half += accel(bodies,tid,N) * dt/2;
    bodies[tid].vel = vel_half;
}

template<typename Body_t>
timed<thrust::host_vector<Body_t>>  run_leapfrog_aos(size_t N, size_t threads_per_block, size_t iters)
{
    thrust::device_vector<Body_t> bodies = make_random_bodies<Body_t>(N);
    size_t n_blocks = ceildiv(bodies.size(),threads_per_block);

    auto start = std::chrono::high_resolution_clock::now();
    for(auto _=0; _!=iters; ++_) {
        leapfrog_aos <<< n_blocks, threads_per_block, 0 >>> (thrust::raw_pointer_cast(bodies.data()), bodies.size(), iters);
        hipDeviceSynchronize();
        quitOnCudaError();
    }
    auto end = std::chrono::high_resolution_clock::now();

    thrust::host_vector<Body_t> res(bodies);
    auto elapsed_seconds = std::chrono::duration_cast<seconds>(end - start);
    return {elapsed_seconds, res};
}

template<typename Body_t>
thrust::device_vector<Body_t> make_random_bodies(size_t N)
{
    thrust::device_vector<Body_t> ret;

    std::default_random_engine generator(0xdeadbeef);
    std::uniform_real_distribution<float> position_distribution(-1,1);
    std::uniform_real_distribution<float> mass_distribution(0.1f,5.f);
    auto x = std::bind(position_distribution, generator);
    auto m = std::bind(mass_distribution, generator);

    for(auto _=0;_!=N;++_) {
        ret.push_back(Body_t{{x(), x(), x()}, m(), {0.f, 0.f, 0.f}});
    }

    return ret;
}

timed<thrust::host_vector<Body>> run_leapfrog_aos(size_t N, size_t threads_per_block, size_t iters)
{
    return run_leapfrog_aos<Body>(N,threads_per_block,iters);
}

timed<thrust::host_vector<UnalignedBody>> run_leapfrog_aos_unaligned(size_t N, size_t threads_per_block, size_t iters)
{
    return run_leapfrog_aos<UnalignedBody>(N,threads_per_block,iters);
}